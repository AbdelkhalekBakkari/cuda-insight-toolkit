#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

template <class T, class S>
__global__ void binaryThreshold(S *output, T lower, T upper, S inside, S outside, int N)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      T A = output[idx];
      if ( lower <= A && A <= upper )
      {
      output[idx] = inside;
      }
      else
	{
	output[idx] = outside;
	}
   }
}

template <class T, class S>
__global__ void binaryThreshold(S *output, const T *input, T lower, T upper, S inside, S outside, int N)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      T A = input[idx];
      if ( lower <= A && A <= upper )
      {
      output[idx] = inside;
      }
      else
	{
	output[idx] = outside;
	}
   }
}

template <class T, class S> 
void BinaryThresholdImageKernelFunction(const T* input, S* output, T m_LowerThreshold,
T m_UpperThreshold, S m_InsideValue, S m_OutsideValue, unsigned int N)
{
   // Compute execution configuration
   int blockSize = 128;
   int nBlocks = N/blockSize + (N%blockSize == 0?0:1);
   // Call  kernel
   if (output == input)
     binaryThreshold <<< nBlocks, blockSize >>> (output, m_LowerThreshold, m_UpperThreshold, m_InsideValue, m_OutsideValue, N);
   else
     binaryThreshold <<< nBlocks, blockSize >>> (output, input, m_LowerThreshold, m_UpperThreshold, m_InsideValue, m_OutsideValue, N);

}

// versions we wish to compile
#define THISTYPE float
template void BinaryThresholdImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, THISTYPE m_LowerThreshold, THISTYPE m_UpperThreshold, THISTYPE m_InsideValue, THISTYPE m_OutsideValue, unsigned int N);
#undef THISTYPE

#define THISTYPE int
template void BinaryThresholdImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, THISTYPE m_LowerThreshold, THISTYPE m_UpperThreshold, THISTYPE m_InsideValue, THISTYPE m_OutsideValue, unsigned int N);

#undef THISTYPE

#define THISTYPE short
template void BinaryThresholdImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, THISTYPE m_LowerThreshold, THISTYPE m_UpperThreshold, THISTYPE m_InsideValue, THISTYPE m_OutsideValue, unsigned int N);

#undef THISTYPE

#define THISTYPE char
template void BinaryThresholdImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, THISTYPE m_LowerThreshold, THISTYPE m_UpperThreshold, THISTYPE m_InsideValue, THISTYPE m_OutsideValue, unsigned int N);

#undef THISTYPE

