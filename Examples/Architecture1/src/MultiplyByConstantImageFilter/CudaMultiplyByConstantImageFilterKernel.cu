#include "hip/hip_runtime.h"
/*
 * File Name:    cuda-kernel.cu
 *
 * Author:        Phillip Ward
 * Creation Date: Monday, January 18 2010, 10:00 
 * Last Modified: Wednesday, December 23 2009, 16:35 
 * 
 * File Description:
 *
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>


template <class S>
__global__ void MultiplyByConstantImageKernel(S *output, int N, S C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] *= C;
   }
}

template <class T, class S>
__global__ void MultiplyByConstantImageKernel(S *output, const T *input, int N, T C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] = input[idx] * C;
   }
}

template<class T, class S>
void MultiplyByConstantImageKernelFunction(const T* input, S* output, unsigned int N, T C)
{
   // Compute execution configuration 
   int blockSize = 128;
   int nBlocks = N/blockSize + (N%blockSize == 0?0:1);


   // Call kernal
   if (output == input)
     MultiplyByConstantImageKernel <<< nBlocks, blockSize >>> (output, N, C);
   else
     MultiplyByConstantImageKernel <<< nBlocks, blockSize >>> (output, input, N, C);

}
// versions we wish to compile
#define THISTYPE float
template void MultiplyByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, unsigned int N, THISTYPE C);
#undef THISTYPE
#define THISTYPE int
template void MultiplyByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE short
template void MultiplyByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE char
template void MultiplyByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE *output, unsigned int N, THISTYPE C);
#undef THISTYPE
