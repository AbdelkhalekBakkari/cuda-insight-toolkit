#include "hip/hip_runtime.h"
/*=========================================================================

  Program:   Insight Segmentation & Registration Toolkit
  Module:    CudaDivideByConstantImageFilterKernel.cu
  Language:  CUDA

  Copyright (c) Insight Software Consortium. All rights reserved.
  See ITKCopyright.txt or http://www.itk.org/HTML/Copyright.htm for details.

     This software is distributed WITHOUT ANY WARRANTY; without even
     the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
     PURPOSE.  See the above copyright notices for more information.

=========================================================================*/
/** \class CudaDivideByConstantImageFilterKernel.cu
 * \brief Cuda kernel code
 * \author Phillip Ward, Luke Parkinson, Daniel Micevski, Christopher
 * Share, Victorian Partnership for Advanced Computing (VPAC). 
 * Richard Beare, Monash University
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

template <class S>
__global__ void DivideByConstantImageKernel(S *output, int N, S C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] /= C;
   }
}

template <class T, class S>
__global__ void DivideByConstantImageKernel(S *output, const T *input, int N, T C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] = input[idx] / C;
   }
}

template<class T, class S>
void DivideByConstantImageKernelFunction(const T* input, S* output, unsigned int N, T C)
{
   // Compute execution configuration 
   int blockSize = 128;
   int nBlocks = N/blockSize + (N%blockSize == 0?0:1);


   // Call kernel
   if (output == input)
     DivideByConstantImageKernel <<< nBlocks, blockSize >>> (output, N, C);
   else
     DivideByConstantImageKernel <<< nBlocks, blockSize >>> (output, input, N, C);

}
// versions we wish to compile
#define THISTYPE float
template void DivideByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE
 * output, unsigned int N, THISTYPE C);
#undef THISTYPE
#define THISTYPE int
template void DivideByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE
 *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE short
template void DivideByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE
 *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE unsigned char
template void DivideByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE
 *output, unsigned int N, THISTYPE C);
#undef THISTYPE
