#include "hip/hip_runtime.h"
/*=========================================================================

  Program:   Insight Segmentation & Registration Toolkit
  Module:    CudaMeanImageFilterKernel.cu
  Language:  CUDA

  Copyright (c) Insight Software Consortium. All rights reserved.
  See ITKCopyright.txt or http://www.itk.org/HTML/Copyright.htm for details.

     This software is distributed WITHOUT ANY WARRANTY; without even
     the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
     PURPOSE.  See the above copyright notices for more information.

=========================================================================*/
/** \class CudaMeanImageFilterKernel.cu
 * \brief Cuda kernel code
 * \author Phillip Ward, Luke Parkinson, Daniel Micevski, Christopher
 * Share, Victorian Partnership for Advanced Computing (VPAC). 
 * Richard Beare, Monash University
 */

#include "EclipseCompat.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "CudaNeighborhoodFunctions.cu"

// Pointer to shared memory window
extern __shared__ float sharedMem[];
// Pointer to 2D Texture
texture<float, 2> texRef2D;
// Pointer to 3D Texture
texture<float, 3> texRef3D;

template<class T>
__global__ void CudaMeanImageFilterKernelShared2D(T *output, int2 imageDim,
		int2 radius, int N) {

	// Compute pixel coordinates of thread
	int2 pixel = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y
			* blockDim.y) + threadIdx.y);

	// Compute window size for shared memory
	int2 window = make_int2((2 * radius.x) + blockDim.x, (2 * radius.y)
			+ blockDim.y);

	// Populate shared memory window
	PopulateSharedMem2D(sharedMem, texRef2D, imageDim, radius, pixel, window);

	T mean = 0;
	// Sync threads to ensure window is completely populated
	__syncthreads();

	// Returns each value in neighborhood
	for (int j = 0; j <= 2 * radius.y; ++j) {
		for (int i = 0; i <= 2 * radius.x; ++i) {
			// Critical section here
			mean += GetSharedMemValue2D(sharedMem, i, j, imageDim, radius,
					pixel, window);
		}
	}

	// Sync threads before doing writes to global
	if (pixel.y < imageDim.y && pixel.x < imageDim.x) {
		// Write value to output here
		output[pixel.y * imageDim.x + pixel.x] = mean / ((2 * radius.x + 1)
				* (2 * radius.y + 1));
	}
}

template<class T>
__global__ void CudaMeanImageFilterKernelGlobal2D(T *output, int2 imageDim,
		int2 radius, int N, int offset) {

	// Compute threads linear position
	int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

	if (idx < N) {
		// Compute pixel coordinates of thread
		int2 pixel = make_int2(idx % imageDim.x, idx / imageDim.x);
		T mean = 0;

		// Returns each value in neighborhood
		for (int j = 0; j <= 2 * radius.y; j++) {
			for (int i = 0; i <= 2 * radius.x; i++) {
				// Add critical section here
				mean += GetGlobalValue2D(texRef2D, i, j, imageDim, radius,
						pixel);
			}
		}

		// Sync before writing back to global
		__syncthreads();

		// Write output
		output[idx] = mean / ((2 * radius.x + 1) * (2 * radius.y + 1));
	}
}

template<class T>
__global__ void CudaMeanImageFilterKernelGlobal3D(T *output, int3 imageDim,
		int3 radius, int N, int offset) {

	// Compute threads linear position
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

	if (idx < N) {

		// Compute pixel coordinates of thread
		int3 pixel = make_int3(0, 0, 0);
		pixel.x = idx % imageDim.x;
		pixel.y = (idx / imageDim.x) % imageDim.y;
		pixel.z = idx / (imageDim.y * imageDim.x);

		T mean = 0;

		// Returns each value in neighborhood
		for (int k = 0; k <= 2 * radius.z; k++) {
			for (int j = 0; j <= 2 * radius.y; j++) {
				for (int i = 0; i <= 2 * radius.x; i++) {
					// Add critical section here
					mean += GetGlobalValue3D(texRef3D, i, j, k, imageDim,
							radius, pixel);
				}
			}
		}

		// Sync before writing back to global
		__syncthreads();

		// Write output
		output[idx] = mean / ((2 * radius.x + 1) * (2 * radius.y + 1) * (2
				* radius.z + 1));
	}
}
template <class T, class S>
void CudaMeanImageFilterKernelFunction(const T* input, S *output,
		unsigned int imageDimX, unsigned int imageDimY, unsigned int imageDimZ,
		unsigned int radiusX, unsigned int radiusY, unsigned int radiusZ,
		unsigned int N) 
{

	// Get device properties to compute block and grid size later
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);

	// 3D Image
	if (imageDimZ > 1) {
		int3 radius = make_int3(radiusX, radiusY, radiusZ);
		int3 imageDim = make_int3(imageDimX, imageDimY, imageDimZ);

		// Allocate Cuda Array
		hipArray *texArray = 0;
		hipChannelFormatDesc cf = hipCreateChannelDesc<T> ();
		hipExtent const ext = { imageDim.x, imageDim.y, imageDim.z };
		CUDA_SAFE_CALL(hipMalloc3DArray(&texArray, &cf, ext));
		CUT_CHECK_ERROR("Malloc 3D Array Failed\n");

		// Bind to Texture
		CUDA_SAFE_CALL(hipBindTextureToArray(texRef3D, texArray, cf));
		CUT_CHECK_ERROR("Bind Texture To Array Failed\n");

		// Copy Linear Device Memory into Cuda Array
		hipMemcpy3DParms copyParams = { 0 };
		copyParams.srcPtr = make_hipPitchedPtr(const_cast<T *> (input),
				imageDim.x * sizeof(T), imageDim.x, imageDim.y);
		copyParams.dstArray = texArray;
		copyParams.kind = hipMemcpyDeviceToDevice;
		copyParams.extent = ext;
		CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
		CUT_CHECK_ERROR("Memcpy Device -> Array Failed\n");

		// specify mutable texture reference parameters
		texRef3D.normalized = 0;
		texRef3D.filterMode = hipFilterModePoint;
		texRef3D.addressMode[0] = hipAddressModeClamp;
		texRef3D.addressMode[1] = hipAddressModeClamp;
		texRef3D.addressMode[2] = hipAddressModeClamp;

		// Calculate block size based on register limit
		int blockSize = 64;//devProp.maxThreadsPerBlock;

		while (blockSize * 19 > devProp.regsPerBlock) {
			blockSize /= 2;
		}

		// Calculate Grid Size and Kernel Passes Required
		int nBlocks = N / blockSize + (N % blockSize == 0 ? 0 : 1);
		int runs = nBlocks / (devProp.maxGridSize[0]) + (nBlocks
				% (devProp.maxGridSize[0]) == 0 ? 0 : 1);

		int i = 1;
		for (; i * 1000 < radius.x * radius.y * radius.z; ++i)
			;
		runs *= i;

		nBlocks /= runs;

		// Execute Kernel Passes
		for (int i = 0; i < runs; i++) {
			int offset = i * nBlocks * blockSize;
			CudaMeanImageFilterKernelGlobal3D <<< nBlocks, blockSize >>>
			(output, imageDim, radius, (int)N, offset);
			hipDeviceSynchronize();
		}

		//printf("Kernel Error: %s\n", hipGetErrorString(hipGetLastError()));

		// Free Array and Unbind Texture
		hipFreeArray(texArray);
		hipUnbindTexture(texRef3D);

		// Return pointer to the output
	}
	// 2D Image
	else {

		int2 radius = make_int2(radiusX, radiusY);
		int2 imageDim2 = make_int2(imageDimX, imageDimY);

		// set up the CUDA array
		hipChannelFormatDesc cf = hipCreateChannelDesc<T> ();
		hipArray *texArray = 0;
		hipMallocArray(&texArray, &cf, imageDim2.x, imageDim2.y);
		hipMemcpyToArray(texArray, 0, 0, input, sizeof(T) * N,
				hipMemcpyDeviceToDevice);

		// specify mutable texture reference parameters
		texRef2D.normalized = 0;
		texRef2D.filterMode = hipFilterModePoint;
		texRef2D.addressMode[0] = hipAddressModeClamp;
		texRef2D.addressMode[1] = hipAddressModeClamp;

		// bind texture reference to array
		hipBindTextureToArray(texRef2D, texArray);

		// Calculate block size based on Maximum Registers per Block
		int blockSize = devProp.maxThreadsPerBlock;
		while (blockSize * 19 > devProp.regsPerBlock) {
			blockSize /= 2;
		}

		// Calculate Grid Size and Kernel Passes Required
		int nBlocks = N / blockSize + (N % blockSize == 0 ? 0 : 1);
		int runs = nBlocks / devProp.maxGridSize[0] + (nBlocks
				% devProp.maxGridSize[0] == 0 ? 0 : 1);

		int i = 1;
		for (; i * 1000 < radius.x * radius.y; ++i)
			;
		runs *= i;
		nBlocks /= runs;

		// Execute Kernel Passes
		for (int i = 0; i < runs; i++) {
			int offset = i * nBlocks * blockSize;
			CudaMeanImageFilterKernelGlobal2D <<< nBlocks, blockSize >>> (output, imageDim2, radius, (int)N, offset);
			hipDeviceSynchronize();
		}

		//printf("Kernel Error: %s\n", hipGetErrorString(hipGetLastError()));

		hipFreeArray(texArray);
		hipUnbindTexture(texRef2D);

		// Return pointer to the output
	}
}

#define THISTYPE float
template void CudaMeanImageFilterKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input1, THISTYPE * output, 
							 unsigned int imageDimX, unsigned int imageDimY, 
							 unsigned int imageDimZ,
							 unsigned int radiusX, unsigned int radiusY, 
							 unsigned int radiusZ,
							 unsigned int N);
#undef THISTYPE
#define THISTYPE int
template void CudaMeanImageFilterKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input1, THISTYPE * output, 
							 unsigned int imageDimX, unsigned int imageDimY, 
							 unsigned int imageDimZ,
							 unsigned int radiusX, unsigned int radiusY, 
							 unsigned int radiusZ,
							 unsigned int N);
#undef THISTYPE

#define THISTYPE short
template void CudaMeanImageFilterKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input1, THISTYPE * output, 
							 unsigned int imageDimX, unsigned int imageDimY, 
							 unsigned int imageDimZ,
							 unsigned int radiusX, unsigned int radiusY, 
							 unsigned int radiusZ,
							 unsigned int N);
#undef THISTYPE

#define THISTYPE unsigned char
template void CudaMeanImageFilterKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input1, THISTYPE * output, 
							 unsigned int imageDimX, unsigned int imageDimY, 
							 unsigned int imageDimZ,
							 unsigned int radiusX, unsigned int radiusY, 
							 unsigned int radiusZ,
							 unsigned int N);
#undef THISTYPE
