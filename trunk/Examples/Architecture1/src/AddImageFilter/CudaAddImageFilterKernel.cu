#include "hip/hip_runtime.h"
/*=========================================================================

  Program:   Insight Segmentation & Registration Toolkit
  Module:    CudaAddImageFilterKernel.cu
  Language:  CUDA

  Copyright (c) Insight Software Consortium. All rights reserved.
  See ITKCopyright.txt or http://www.itk.org/HTML/Copyright.htm for details.

     This software is distributed WITHOUT ANY WARRANTY; without even
     the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
     PURPOSE.  See the above copyright notices for more information.

=========================================================================*/
/** \class CudaAddImageFilterKernel.cu
 * \brief Cuda kernel code
 * \author Phillip Ward, Luke Parkinson, Daniel Micevski, Christopher
 * Share, Victorian Partnership for Advanced Computing (VPAC). 
 * Richard Beare, Monash University
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

// #include "thrust/functional.h"
// #include "thrust/transform.h"

template <class T, class S>
__global__ void AddImageKernel(T *output, const S *input, int N)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) 
    {
    output[idx] += input[idx];
    }
}

template <class T, class S>
__global__ void AddImageKernel(T *output, const S *input1, const S* input2, int N)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
     {
     output[idx] = input1[idx] + input2[idx];
     }
}

template <class T, class S>
void AddImageKernelFunction(const T* input1, const T* input2, S* output, unsigned int N)
{


    // Compute execution configuration 
    int blockSize = 128;
    int nBlocks = N/blockSize + (N%blockSize == 0?0:1);

    // Call kernel
    if (output == input1)
      AddImageKernel <<< nBlocks, blockSize >>> (output, input2, N);
    else
      AddImageKernel <<< nBlocks, blockSize >>> (output, input1, input2, N);
}



// template <class T, class S>
// void AddImageKernelFunction(const T* input1, const T* input2, S* output, unsigned int N)
// {
//   if (input1 == output)
//     {
//     // not sure if this makes any difference
//     thrust::device_ptr<const T> i2(input2);
//     thrust::device_ptr<S> o1(output);
//     thrust::transform(o1, o1 + N, i2, o1, thrust::plus<S>());
//     }
//   else
//     {
//     thrust::device_ptr<const T> i1(input1);
//     thrust::device_ptr<const T> i2(input2);
//     thrust::device_ptr<S> o1(output);
//     thrust::transform(i1, i1 + N, i2, o1, thrust::plus<S>());
//     }

// }


// versions we wish to compile
#define THISTYPE float
template void AddImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input1, const THISTYPE * input2, THISTYPE * output, unsigned int N);
#undef THISTYPE
#define THISTYPE int
template void AddImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input1, const THISTYPE * input2, THISTYPE *output, unsigned int N);
#undef THISTYPE

#define THISTYPE short
template void AddImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input1, const THISTYPE * input2, THISTYPE *output, unsigned int N);
#undef THISTYPE

#define THISTYPE unsigned char
template void AddImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input1, const THISTYPE * input2,  THISTYPE *output, unsigned int N);
#undef THISTYPE
