#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <cutil.h>
#include "CudaStatisticsImageFilterKernel.h"

template <class T>
__global__ void StatisticsImageKernel(T *output, float* maxImage,
				      float* minImage, float* sumImage, 
				      float* sumOfSquaresImage) 
{

  int idx = threadIdx.x;

  for (float offset = blockDim.x / 2 ; offset >= 1; offset /= 2)
    {
    if (idx < offset)  {
    int next = (int)(idx + offset);
    maxImage[idx] = (maxImage[idx] > maxImage[next] ? maxImage[idx] : maxImage[next]);
    minImage[idx] = (minImage[idx] < minImage[next] ? minImage[idx] : minImage[next]);
    sumImage[idx] += sumImage[next];
    sumOfSquaresImage[idx] += sumOfSquaresImage[next];
    }
    __syncthreads();
    }

  if (blockIdx.x == 0)
    {
    for (float offset = gridDim.x / 2 ; offset >= 1; offset /= 2)
      {
      if (idx < offset)  {
      int next = (int)(idx + offset) * blockDim.x;
      maxImage[idx * blockDim.x] = (maxImage[idx * blockDim.x] > maxImage[next] ? maxImage[idx * blockDim.x] : maxImage[next]);
      minImage[idx * blockDim.x] = (minImage[idx * blockDim.x] < minImage[next] ? minImage[idx * blockDim.x] : minImage[next]);
      sumImage[idx * blockDim.x] += sumImage[next];
      sumOfSquaresImage[idx * blockDim.x] += sumOfSquaresImage[next];
      }
      offset /= 2;
      __syncthreads();
      }
    }
}
template <class T>
__global__ void StatisticsSquareImageKernel(T* sumOfSquaresImage, int N) 
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N)
    {
    sumOfSquaresImage[idx] *= sumOfSquaresImage[idx];
    }
}

template <class T> 
void StatisticsImageKernelFunction(const T* input, T* output,
				   StatisticsStruct * stats, unsigned int N) 
{

  float *maxImage, *minImage, *sumImage, *sumOfSquaresImage;

  hipMalloc( &maxImage, sizeof(float) * N);
  hipMalloc( &minImage, sizeof(float) * N);
  hipMalloc( &sumImage, sizeof(float) * N);
  hipMalloc( &sumOfSquaresImage, sizeof(float) * N);

  hipMemcpy(maxImage, output, sizeof(float) * N, hipMemcpyDeviceToDevice);
  hipMemcpy(minImage, output, sizeof(float) * N, hipMemcpyDeviceToDevice);
  hipMemcpy(sumImage, output, sizeof(float) * N, hipMemcpyDeviceToDevice);
  hipMemcpy(sumOfSquaresImage, output, sizeof(float) * N,
	     hipMemcpyDeviceToDevice);

  // Compute execution configuration
  int blockSize = 128;
  int nBlocks = N / blockSize + (N % blockSize == 0 ? 0 : 1);

  // Call kernel
  StatisticsSquareImageKernel <<< nBlocks, blockSize >>> (sumOfSquaresImage, N);
  hipDeviceSynchronize();
  StatisticsImageKernel <<< nBlocks, blockSize >>> (output, maxImage, minImage, sumImage, sumOfSquaresImage);

  float max, min, sum, sumOfSquares;

  hipMemcpy(&sum, sumImage, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&max, maxImage, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&min, minImage, sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(&sumOfSquares, sumOfSquaresImage, sizeof(float),
	     hipMemcpyDeviceToHost);

  stats->Maximum = max;
  stats->Minimum = min;
  stats->Sum = sum;
  stats->Mean = sum / N;
  stats->Variance = (sumOfSquares - (sum * sum / N)) / (N - 1);
  stats->Sigma = sqrtf(stats->Variance);

  // Clean up
  hipFree(maxImage);
  hipFree(minImage);
  hipFree(sumImage);
  hipFree(sumOfSquaresImage);

}
// versions we wish to compile
#define THISTYPE float
template void BinaryThresholdImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, StatisticsStruct * stats, unsigned int N);
#undef THISTYPE

#define THISTYPE int
template void BinaryThresholdImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, StatisticsStruct * stats, unsigned int N);

#undef THISTYPE

#define THISTYPE short
template void BinaryThresholdImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, StatisticsStruct * stats, unsigned int N);

#undef THISTYPE

#define THISTYPE char
template void BinaryThresholdImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, StatisticsStruct * stats, unsigned int N);

#undef THISTYPE
