#include "hip/hip_runtime.h"
/*=========================================================================

  Program:   Insight Segmentation & Registration Toolkit
  Module:    CudaSubtractConstantFromImageFilterKernel.cu
  Language:  CUDA

  Copyright (c) Insight Software Consortium. All rights reserved.
  See ITKCopyright.txt or http://www.itk.org/HTML/Copyright.htm for details.

     This software is distributed WITHOUT ANY WARRANTY; without even
     the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
     PURPOSE.  See the above copyright notices for more information.

=========================================================================*/
/** \class CudaSubtractConstantFromImageFilterKernel.cu
 * \brief Cuda kernel code
 * \author Phillip Ward, Luke Parkinson, Daniel Micevski, Christopher
 * Share, Victorian Partnership for Advanced Computing (VPAC). 
 * Richard Beare, Monash University
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

template <class S>
__global__ void SubtractConstantFromImageKernel(S *output, int N, S C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] -= C;
   }
}

template <class T, class S>
__global__ void SubtractConstantFromImageKernel(S *output, const T *input, int N, T C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] = input[idx] - C;
   }
}

template<class T, class S>
void SubtractConstantFromImageKernelFunction(const T* input, S* output, unsigned int N, T C)
{
   // Compute execution configuration 
   int blockSize = 128;
   int nBlocks = N/blockSize + (N%blockSize == 0?0:1);


   // Call kernel
   if (output == input)
     SubtractConstantFromImageKernel <<< nBlocks, blockSize >>> (output, N, C);
   else
     SubtractConstantFromImageKernel <<< nBlocks, blockSize >>> (output, input, N, C);

}

// versions we wish to compile
#define THISTYPE float
template void SubtractConstantFromImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, unsigned int N, THISTYPE C);
#undef THISTYPE
#define THISTYPE int
template void SubtractConstantFromImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE short
template void SubtractConstantFromImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE unsigned char
template void SubtractConstantFromImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE *output, unsigned int N, THISTYPE C);
#undef THISTYPE

