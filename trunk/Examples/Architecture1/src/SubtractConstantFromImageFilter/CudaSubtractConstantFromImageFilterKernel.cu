#include "hip/hip_runtime.h"
/*
 * File Name:    cuda-kernel.cu
 *
 * Author:        Phillip Ward
 * Creation Date: Monday, January 18 2010, 10:00 
 * Last Modified: Wednesday, December 23 2009, 16:35 
 * 
 * File Description:
 *
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

template <class S>
__global__ void SubtractConstantFromImageKernel(S *output, int N, S C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] -= C;
   }
}

template <class T, class S>
__global__ void SubtractConstantFromImageKernel(S *output, const T *input, int N, T C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] = input[idx] - C;
   }
}

template<class T, class S>
void SubtractConstantFromImageKernelFunction(const T* input, S* output, unsigned int N, T C)
{
   // Compute execution configuration 
   int blockSize = 128;
   int nBlocks = N/blockSize + (N%blockSize == 0?0:1);


   // Call kernal
   if (output == input)
     SubtractConstantFromImageKernel <<< nBlocks, blockSize >>> (output, N, C);
   else
     SubtractConstantFromImageKernel <<< nBlocks, blockSize >>> (output, input, N, C);

}

// versions we wish to compile
#define THISTYPE float
template void SubtractConstantFromImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE * output, unsigned int N, THISTYPE C);
#undef THISTYPE
#define THISTYPE int
template void SubtractConstantFromImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE short
template void SubtractConstantFromImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE char
template void SubtractConstantFromImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE *output, unsigned int N, THISTYPE C);
#undef THISTYPE

