#include "hip/hip_runtime.h"
/*
 * File Name:    Cuda Texture Functions
 *
 * Author:        Phillip Ward
 * Creation Date: Monday, January 18 2010, 10:00 
 * Last Modified: Wednesday, February 23 2010, 16:35
 * 
 * File Description:
 *
 */
#include "EclipseCompat.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

void copy3DHostToArray(float *_src, hipArray *_dst, hipExtent copy_extent, hipPos src_offset, int3 imageDim)
{
 hipMemcpy3DParms copyParams = {0};
 float *h_source = _src + src_offset.x + src_offset.y*imageDim.x + src_offset.z*imageDim.x*imageDim.y;
 copyParams.srcPtr = make_hipPitchedPtr((void*)h_source, imageDim.x*sizeof(float), imageDim.x, imageDim.y);
 copyParams.dstArray = _dst;
 copyParams.kind = hipMemcpyHostToDevice;
 copyParams.extent = copy_extent;

 CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
 CUT_CHECK_ERROR("Host -> Array Memcpy failed\n");
}

void copy3DDeviceToArray(float *_src, hipArray *_dst, hipExtent copy_extent, hipPos src_offset, int3 imageDim)
{
 hipMemcpy3DParms copyParams = {0};
 float *d_source = _src + src_offset.x + src_offset.y*imageDim.x + src_offset.z*imageDim.x*imageDim.y;
 copyParams.srcPtr = make_hipPitchedPtr((void*)d_source, imageDim.x*sizeof(float), imageDim.x, imageDim.y);
 copyParams.dstArray = _dst;
 copyParams.kind = hipMemcpyDeviceToDevice;
 copyParams.extent = copy_extent;

 CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
 CUT_CHECK_ERROR("Device -> Array Memcpy failed\n");
}

void copy3DMemToArray(hipPitchedPtr _src, hipArray *_dst, int3 imageDim)
{
 hipMemcpy3DParms copyParams = {0};
 copyParams.srcPtr =  _src;
 copyParams.dstArray = _dst;
 copyParams.kind = hipMemcpyDeviceToDevice;
 copyParams.extent = make_hipExtent(imageDim.x, imageDim.y, imageDim.z);

 CUDA_SAFE_CALL(hipMemcpy3D(&copyParams));
 CUT_CHECK_ERROR("Mem -> Array Memcpy failed\n");
}
