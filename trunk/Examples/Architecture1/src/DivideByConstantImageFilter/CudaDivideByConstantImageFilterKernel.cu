#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>

template <class S>
__global__ void DivideByConstantImageKernel(S *output, int N, S C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] /= C;
   }
}

template <class T, class S>
__global__ void DivideByConstantImageKernel(S *output, const T *input, int N, T C)
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   if (idx<N) 
   {
      output[idx] = input[idx] / C;
   }
}

template<class T, class S>
void DivideByConstantImageKernelFunction(const T* input, S* output, unsigned int N, T C)
{
   // Compute execution configuration 
   int blockSize = 128;
   int nBlocks = N/blockSize + (N%blockSize == 0?0:1);


   // Call kernel
   if (output == input)
     DivideByConstantImageKernel <<< nBlocks, blockSize >>> (output, N, C);
   else
     DivideByConstantImageKernel <<< nBlocks, blockSize >>> (output, input, N, C);

}
// versions we wish to compile
#define THISTYPE float
template void DivideByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE
 * output, unsigned int N, THISTYPE C);
#undef THISTYPE
#define THISTYPE int
template void DivideByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE
 *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE short
template void DivideByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE
 *output, unsigned int N, THISTYPE C);
#undef THISTYPE

#define THISTYPE unsigned char
template void DivideByConstantImageKernelFunction<THISTYPE, THISTYPE>(const THISTYPE * input, THISTYPE
 *output, unsigned int N, THISTYPE C);
#undef THISTYPE
